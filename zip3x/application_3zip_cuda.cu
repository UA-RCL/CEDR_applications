#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <cmath>
#include <complex>
#include "libcedr.h"
#include <time.h>
#include <mutex>
#include <inttypes.h>
#include <cstring>
#include <hip/hip_runtime.h>

using namespace std;
#define SEC2NANOSEC 1000000000


// Function for launching ZIP on GPU
__global__ void vector_mult(const cedr_cmplx_flt_type* x, const cedr_cmplx_flt_type* y, cedr_cmplx_flt_type* z, int len) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  cedr_re_flt_type r1=x[id].re, r2=y[id].re;
  cedr_re_flt_type i1=x[id].im, i2=y[id].im;
  if (id < len) {
    z[id].re = r1*r2 - i1*i2;
    z[id].im = r1*i2 + r2*i1;
  }
}

int main(int argc, char** argv) {
  #if defined(TIME_CAPTURE)
  const char* filename_time = "results_3zip_cuda.csv";  // Replace with your desired filename
  // Open the file in append mode
  FILE* file = fopen(filename_time, "w");
  if (file == NULL) {
    printf("Failed to open the file.\n");
    return 1;
  }
  fprintf(file, "Size\tAllocation Time\tZIP Execution Time\tDeallocation Time\n");
  struct timespec start_time{};
  struct timespec end_time{};
  long long start, end;
  long long total_time;
  #endif
  int M; // Repeated experiments
  if (argc == 2){
    M = atoi(argv[1]);
  } else{
    M = 100;
  }
  cout << "[APP] Launched app main function with M=" << M << "!" << endl;

  // Allocate input and output memories and generate input
  cedr_cmplx_flt_type *ref_A0, *ref_A1;
  cedr_cmplx_flt_type *ref_B0, *ref_B1;
  cedr_cmplx_flt_type *ref_C0, *ref_C1;
  cedr_cmplx_flt_type *ref_D;
  cedr_cmplx_flt_type *dev_A0, *dev_A1;
  cedr_cmplx_flt_type *dev_B0, *dev_B1;
  cedr_cmplx_flt_type *dev_C0, *dev_C1;
  cedr_cmplx_flt_type *dev_D;


  // CEDR_ZIP API call argument setup
  size_t ZIP_SIZE;  // = N;

  for (size_t N = 64; N <= 65536*2; N=N*2){
    cout << "[APP] Running experiments for size:" << N << endl;
    #if defined(TIME_CAPTURE)
    fprintf(file, "%ld\t", N);
    #endif
    ZIP_SIZE = N;

    // GPU stuff
    const int threadsPerBlock = 512;
    const int blocksPerGrid = (ZIP_SIZE + threadsPerBlock - 1) / threadsPerBlock;

    #if defined(TIME_CAPTURE)
    clock_gettime(CLOCK_MONOTONIC_RAW, &start_time);
    #endif

    ref_A0 = new cedr_cmplx_flt_type[N];
    ref_A1 = new cedr_cmplx_flt_type[N];
    ref_B0 = new cedr_cmplx_flt_type[N];
    ref_B1 = new cedr_cmplx_flt_type[N];
    ref_C0 = new cedr_cmplx_flt_type[N];
    ref_C1 = new cedr_cmplx_flt_type[N];
    ref_D  = new cedr_cmplx_flt_type[N];

    hipMalloc((void**)&dev_A0, N*sizeof(cedr_cmplx_flt_type));
    hipMalloc((void**)&dev_A1, N*sizeof(cedr_cmplx_flt_type));
    hipMalloc((void**)&dev_B0, N*sizeof(cedr_cmplx_flt_type));
    hipMalloc((void**)&dev_B1, N*sizeof(cedr_cmplx_flt_type));
    hipMalloc((void**)&dev_C0, N*sizeof(cedr_cmplx_flt_type));
    hipMalloc((void**)&dev_C1, N*sizeof(cedr_cmplx_flt_type));
    hipMalloc((void**)&dev_D, N*sizeof(cedr_cmplx_flt_type));

    #if defined(TIME_CAPTURE)
    clock_gettime(CLOCK_MONOTONIC_RAW, &end_time);
    end = end_time.tv_nsec + end_time.tv_sec*SEC2NANOSEC;
    start = start_time.tv_nsec + start_time.tv_sec*SEC2NANOSEC;
    fprintf(file, "%lld\t", end-start);
    #endif

    for (int i = 0; i < N; i++) {
      ref_A0[i].re = i;
      ref_A0[i].im = i*2;
      ref_A1[i].re = i*3;
      ref_A1[i].im = i*4;
      ref_B0[i].re = i;
      ref_B0[i].im = i*2;
      ref_B1[i].re = i*3;
      ref_B1[i].im = i*4;
      ref_D[i].re  = 0;
      ref_D[i].im  = 0;
    }

    #if defined(TIME_CAPTURE)
      total_time = 0;
    #endif
    for (int ii=0; ii<M; ii++){
      #if defined(TIME_CAPTURE)
        clock_gettime(CLOCK_MONOTONIC_RAW, &start_time);
      #endif

      //#pragma omp parallel sections num_threads(2)
      //{
      //  #pragma omp section
      //  {
          hipMemcpy(dev_A0, ref_A0, N*sizeof(cedr_cmplx_flt_type), hipMemcpyHostToDevice);
          hipMemcpy(dev_B0, ref_B0, N*sizeof(cedr_cmplx_flt_type), hipMemcpyHostToDevice);
          vector_mult<<<blocksPerGrid, threadsPerBlock>>>(dev_A0, dev_B0, dev_C0, ZIP_SIZE);
      //  }
      //  #pragma omp section
      //  {
          hipMemcpy(dev_A1, ref_A1, N*sizeof(cedr_cmplx_flt_type), hipMemcpyHostToDevice);
          hipMemcpy(dev_B1, ref_B1, N*sizeof(cedr_cmplx_flt_type), hipMemcpyHostToDevice);
          vector_mult<<<blocksPerGrid, threadsPerBlock>>>(dev_A1, dev_B1, dev_C1, ZIP_SIZE);
      //  }
      //}
      // Launch 
      vector_mult<<<blocksPerGrid, threadsPerBlock>>>(dev_C0, dev_C1, dev_D, ZIP_SIZE);
      hipMemcpy(ref_D, dev_D, N*sizeof(cedr_cmplx_flt_type), hipMemcpyDeviceToHost);
      //hipDeviceSynchronize();
      #if defined(TIME_CAPTURE)
        clock_gettime(CLOCK_MONOTONIC_RAW, &end_time);
        start = start_time.tv_nsec + start_time.tv_sec * SEC2NANOSEC;
        end = end_time.tv_nsec + end_time.tv_sec * SEC2NANOSEC;
        if (ii >= 0.1 * M){
          total_time += end - start;
        }
      #endif
    }
    #if defined(TIME_CAPTURE)
      fprintf(file, "%lf\t", total_time/(float)(0.9 * M));
    #endif

    
    // Free the allocated memory
    #if defined(TIME_CAPTURE)
      clock_gettime(CLOCK_MONOTONIC_RAW, &start_time);
    #endif
    delete[] ref_A0;
    delete[] ref_A1;
    delete[] ref_B0;
    delete[] ref_B1;
    delete[] ref_C0;
    delete[] ref_C1;
    delete[] ref_D;
    hipFree(dev_A0);
    hipFree(dev_A1);
    hipFree(dev_B0);
    hipFree(dev_B1);
    hipFree(dev_C0);
    hipFree(dev_C1);
    hipFree(dev_D);
    #if defined(TIME_CAPTURE)
      clock_gettime(CLOCK_MONOTONIC_RAW, &end_time);
      start = start_time.tv_nsec + start_time.tv_sec * SEC2NANOSEC;
      end = end_time.tv_nsec + end_time.tv_sec * SEC2NANOSEC;
      total_time = end - start;
      fprintf(file, "%lld\n", total_time);
    #endif
  }

  cout << "[APP] Exiting app main function!" << endl;

  return 0;
}
